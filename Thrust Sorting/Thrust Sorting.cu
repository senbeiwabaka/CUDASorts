#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <iostream>
#include <ctime>

struct Test{
	int key;
	float value;
	double value1;
	char value2;
	
	__host__ __device__
	bool operator<(const Test other) const{
		return key < other.key;
	}
};

void initlialize(thrust::host_vector<Test>& structures){
	thrust::default_random_engine rng;
	thrust::uniform_int_distribution<int> dist(0, 2147483647);
	char alphabet[]={'A','B','C','D','E','F','G','H','I','J','K','L','M',
        'N','O','P','Q','R','S','T','U','V','W','X','Y','Z',
        'a','b','c','d','e','f','g','h','i','j','k','l','m',
        'n','o','p','q','r','s','t','u','v','w','x','y','z'};
	
	for(size_t i = 0; i < structures.size(); i++){
		structures[i].key = (rand()%100)+1;
		structures[i].value = ((float)(rand()%100+1)/101)*100.0;
		structures[i].value1 = ((double)(rand()%100+1)/101)*100.0;
		structures[i].value2 = alphabet[rand()%52];
	}
}

int main(int argc, char** argv){
	srand((unsigned)time(0));
	clock_t start, end;
	
	thrust::host_vector<Test> testH;
	thrust::host_vector<double> doubleH;
	thrust::host_vector<char> charH;
	thrust::device_vector<Test> testD;
	thrust::device_vector<double> doubleD;
	thrust::device_vector<char> charD;
	
	char alphabet[]={'A','B','C','D','E','F','G','H','I','J','K','L','M',
        'N','O','P','Q','R','S','T','U','V','W','X','Y','Z',
        'a','b','c','d','e','f','g','h','i','j','k','l','m',
        'n','o','p','q','r','s','t','u','v','w','x','y','z'};
		
	if(argc == 2){
		if(strncmp(argv[1], "-help", 5) == 0 || strncmp(argv[1], "-h", 2) == 0){
			std::cout << "You can either enter a value for the size of the arrays to be sorted or leave it blank and it will generate arrays to be sorted of size around 20 million values";
			
			std::exit(0);
		}
		else{
			testH = thrust::host_vector<Test>(atoi(argv[1]));
			doubleH = thrust::host_vector<double>(atoi(argv[1]));
			charH = thrust::host_vector<char>(atoi(argv[1]));
			
			testD = thrust::device_vector<Test>(atoi(argv[1]));
			doubleD = thrust::device_vector<double>(atoi(argv[1]));
			charD = thrust::device_vector<char>(atoi(argv[1]));
			
			for(int i = 0; i < atoi(argv[1]); ++i){
				doubleH[i] = ((double)(rand()%100+1)/101)*100.0;
				charH[i] = alphabet[rand()%52];
			}
			
			doubleD = doubleH;
			charD = charH;
			
			initlialize(testH);
		}
	}
	else{
		size_t N = 128;
		testH = thrust::host_vector<Test>(N);
		doubleH = thrust::host_vector<double>(N);
		charH = thrust::host_vector<char>(N);
		
		testD = thrust::device_vector<Test>(N);
		doubleD = thrust::device_vector<double>(N);
		charD = thrust::device_vector<char>(N);
		
		initlialize(testH);
	}

	char selection = '0';
	
	std::cout << "What would you like to sort?" << std::endl;
	std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
	std::cin >> selection;
	
	while(selection != '0'){
		if(selection == '1'){
			thrust::host_vector<char> nonsort(charH.size());
			
			nonsort = charH;
			
			start = clock();
			
			thrust::sort(charH.begin(), charH.end());
			
			end = clock();
			
			std::cout << std::endl << "Time for host sort " << end - start << " milliseconds" << std::endl;
			
			start = clock();
			
			charD = nonsort;
			
			thrust::sort(charD.begin(), charD.end());
			
			thrust::copy(charD.begin(), charD.end(), nonsort.begin());
			
			end = clock();
			
			std::cout << "Time for device sort " << end - start << " milliseconds" << std::endl << std::endl;
			
			std::cout << "What would you like to sort?" << std::endl;
			std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
			std::cin >> selection;
		}
		else if(selection == '2'){
			thrust::host_vector<char> nonsort(doubleH.size());
			nonsort = doubleH;
			
			start = clock();
			
			thrust::sort(doubleH.begin(), doubleH.end());
			
			end = clock();
			
			std::cout << std::endl << "Time for host sort " << end - start << " milliseconds" << std::endl;
			
			start = clock();
			
			doubleD = nonsort;
			
			thrust::sort(doubleD.begin(), doubleD.end());
			
			thrust::copy(doubleD.begin(), doubleD.end(), nonsort.begin());
			
			end = clock();
			
			std::cout << "Time for device sort " << end - start << " milliseconds" << std::endl << std::endl;
			
			std::cout << "What would you like to sort?" << std::endl;
			std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
			std::cin >> selection;
		}
		else if(selection == '3'){
			thrust::host_vector<Test> nonsort(testH.size());
	
			nonsort = testH;
			
			start = clock();
			
			thrust::sort(testH.begin(), testH.end());
			
			end = clock();
			
			std::cout << std::endl << "Time for host sort " << end - start << " milliseconds" << std::endl;
			
			start = clock();
			
			testD = nonsort;
			
			thrust::sort(testD.begin(), testD.end());
			
			thrust::copy(testD.begin(), testD.end(), nonsort.begin());
			
			end = clock();
			
			std::cout << "Time for device sort " << end - start << " milliseconds" << std::endl << std::endl;
			
			std::cout << "What would you like to sort?" << std::endl;
			std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
			std::cin >> selection;
		}
		else{
			std::cout << "What would you like to sort?" << std::endl;
			std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
			std::cin >> selection;
		}
	}

    return 0;
}