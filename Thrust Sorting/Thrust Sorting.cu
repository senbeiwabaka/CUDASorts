#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <iostream>
#include <ctime>

//complex object for testing key-pair sorting
struct Test{
	int key; //key that is being tested for sorting
	float value;
	double value1;
	char value2;
	
	__host__ __device__
	bool operator<(const Test other) const{
		return key < other.key;
	}
};

//random generator for complex object
void initlialize(thrust::host_vector<Test>& structures){
	char alphabet[]={'A','B','C','D','E','F','G','H','I','J','K','L','M',
        'N','O','P','Q','R','S','T','U','V','W','X','Y','Z',
        'a','b','c','d','e','f','g','h','i','j','k','l','m',
        'n','o','p','q','r','s','t','u','v','w','x','y','z'};
	
	for(size_t i = 0; i < structures.size(); i++){
		structures[i].key = (rand()%100)+1;
		structures[i].value = ((float)(rand()%100+1)/101)*100.0;
		structures[i].value1 = ((double)(rand()%100+1)/101)*100.0;
		structures[i].value2 = alphabet[rand()%52];
	}
}

int main(int argc, char** argv){
	srand((unsigned)time(0));
	clock_t start, end;
	
	//variables for sorting host vector (CPU)
	thrust::host_vector<Test> testH;
	thrust::host_vector<double> doubleH;
	thrust::host_vector<char> charH;
	//variables for sorting device vector (GPU)
	thrust::device_vector<Test> testD;
	thrust::device_vector<double> doubleD;
	thrust::device_vector<char> charD;
	
	char alphabet[]={'A','B','C','D','E','F','G','H','I','J','K','L','M',
        'N','O','P','Q','R','S','T','U','V','W','X','Y','Z',
        'a','b','c','d','e','f','g','h','i','j','k','l','m',
        'n','o','p','q','r','s','t','u','v','w','x','y','z'};
		
	//checking arguments
	if(argc == 2){
		if(strncmp(argv[1], "-help", 5) == 0 || strncmp(argv[1], "-h", 2) == 0){
			std::cout << "You can either enter a value for the size of the arrays to be sorted or leave it blank and it will generate arrays to be sorted of size around 20 million values";
			
			std::exit(0);
		}
		//accepts argument for the size to randomly generate for the arrays to sort
		else{
			testH = thrust::host_vector<Test>(atoi(argv[1]));
			doubleH = thrust::host_vector<double>(atoi(argv[1]));
			charH = thrust::host_vector<char>(atoi(argv[1]));
			
			testD = thrust::device_vector<Test>(atoi(argv[1]));
			doubleD = thrust::device_vector<double>(atoi(argv[1]));
			charD = thrust::device_vector<char>(atoi(argv[1]));
			
			for(int i = 0; i < atoi(argv[1]); ++i){
				doubleH[i] = ((double)(rand()%100+1)/101)*100.0;
				charH[i] = alphabet[rand()%52];
			}
			
			doubleD = doubleH;
			charD = charH;
			
			initlialize(testH);
		}
	}
	//sets the size to 128 then generates materials for the arrays.
	else{
		size_t N = 128;
		testH = thrust::host_vector<Test>(N);
		doubleH = thrust::host_vector<double>(N);
		charH = thrust::host_vector<char>(N);
		
		testD = thrust::device_vector<Test>(N);
		doubleD = thrust::device_vector<double>(N);
		charD = thrust::device_vector<char>(N);
		
		for(int i = 0; i < N; ++i){
				doubleH[i] = ((double)(rand()%100+1)/101)*100.0;
				charH[i] = alphabet[rand()%52];
		}
		
		doubleD = doubleH;
		charD = charH;
		
		initlialize(testH);
	}

	char selection = '0';
	
	std::cout << "What would you like to sort?" << std::endl;
	std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
	std::cin >> selection;
	
	//loop so you can continue to choose a data type to sort and see timings
	while(selection != '0'){
		if(selection == '1'){
			thrust::host_vector<char> nonsort(charH.size());
			
			nonsort = charH;
			
			start = clock();
			
			thrust::sort(charH.begin(), charH.end());
			
			end = clock();
			
			for(int i = 0; i < 10; ++i){
				std::cout << charH[i] << " ";
			}
			
			std::cout << std::endl;
			
			std::cout << std::endl << "Time for host sort " << end - start << " milliseconds" << std::endl;
			
			start = clock();
			
			charD = nonsort;
			
			thrust::sort(charD.begin(), charD.end());
			
			hipDeviceSynchronize();
			
			thrust::copy(charD.begin(), charD.end(), nonsort.begin());
			
			end = clock();
			
			for(int i = 0; i < 10; ++i){
				std::cout << nonsort[i] << " ";
			}
			
			std::cout << std::endl;
			
			std::cout << "Time for device sort " << end - start << " milliseconds" << std::endl << std::endl;
			
			std::cout << "What would you like to sort?" << std::endl;
			std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
			std::cin >> selection;
		}
		else if(selection == '2'){
			thrust::host_vector<char> nonsort(doubleH.size());
			nonsort = doubleH;
			
			start = clock();
			
			thrust::sort(doubleH.begin(), doubleH.end());
			
			end = clock();
			
			std::cout << std::endl << "Time for host sort " << end - start << " milliseconds" << std::endl;
			
			start = clock();
			
			doubleD = nonsort;
			
			thrust::sort(doubleD.begin(), doubleD.end());
			
			hipDeviceSynchronize();
			
			thrust::copy(doubleD.begin(), doubleD.end(), nonsort.begin());
			
			end = clock();
			
			std::cout << "Time for device sort " << end - start << " milliseconds" << std::endl << std::endl;
			
			std::cout << "What would you like to sort?" << std::endl;
			std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
			std::cin >> selection;
		}
		else if(selection == '3'){
			thrust::host_vector<Test> nonsort(testH.size());
	
			nonsort = testH;
			
			start = clock();
			
			thrust::sort(testH.begin(), testH.end());
			
			end = clock();
			
			std::cout << std::endl << "Time for host sort " << end - start << " milliseconds" << std::endl;
			
			start = clock();
			
			testD = nonsort;
			
			thrust::sort(testD.begin(), testD.end());
			
			hipDeviceSynchronize();
			
			thrust::copy(testD.begin(), testD.end(), nonsort.begin());
			
			end = clock();
			
			std::cout << "Time for device sort " << end - start << " milliseconds" << std::endl << std::endl;
			
			std::cout << "What would you like to sort?" << std::endl;
			std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
			std::cin >> selection;
		}
		else{
			std::cout << "What would you like to sort?" << std::endl;
			std::cout << "1 char \n2 double \n3 complex object \n0 quit?" << std::endl;
			std::cin >> selection;
		}
	}

    return 0;
}