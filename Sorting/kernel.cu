#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

#include <iostream>

using namespace std;

//Odd-even Bubble Sort
template <class T>
__global__ void BubbleKernel(int size, T* arr){
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	for(int i = 0; i < size; i++)
	{
	  //check if the thread id number is even
	  if((id % 2) == 0 && id + 1 < size )
	    {
	      if(arr[id] > arr[id + 1])
	        {
	          T temp = arr[id + 1];
	          arr[id + 1] = arr[id];
	          arr[id] = temp;
	        }
	    }
	
	 __syncthreads();
	
	  //check if the thread id number is odd
	  if((id % 2) != 0 && id + 1 < size )
	    {
	    if(arr[id] > arr[id + 1])
	      {
	         T temp = arr[id + 1];
	         arr[id + 1] =arr[id];
	         arr[id] = temp;
	      }
	   }
	__syncthreads();
	
	}
}

// Puts the array back together
template <class T>
__device__ void Merge(T* arr, T* results, int l, int r, int u){
	int i,j,k;
	i=l; j=r; k=l;
	while (i<r && j<u) { 
	  if (arr[i]<=arr[j]) {
		  results[k]=arr[i]; 
		  i++;} 
	  else {
		  results[k]=arr[j]; 
		  j++;}
	  k++;
	}
	
	while (i<r){ 
	  results[k]=arr[i]; i++; k++;
	}
	
	while (j<u){ 
	  results[k]=arr[j]; j++; k++;
	}
	for (k=0; k<u; k++){ 
	  arr[k]=results[k]; 
	}
}

//Iterative Merge Sort
template <class T>
__global__ void IterMergeKernel(int size, T* arr){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int k,u,i;

	T* results = new T[size];
    
    k = 1;
    while(k < size)
    {
        i = 1;
        while(i+k < size)
        {
            u = i+k*2;
            if(u > size)
            {
                u = size+1;
            }
            Merge(arr, results, i, i+k, u);
            i = i+k*2;
			__syncthreads();
        }
        k = k*2;
        __syncthreads();
    }
}

//create the memory on the device and copy the values over
//will also return the error if there is one (being not being able to allocate memory or whatever else)
template <class T>
void createCUDAMem(T*& arr, int size, T*& hipArray){
	hipError_t error;

	error = hipMalloc((void**)&hipArray, sizeof(T) * size);

	if(error != hipSuccess){
		cout << "Cuda Error: " << hipGetErrorString(error);
	}

	error = hipMemcpy(hipArray, arr, sizeof(T) * size, hipMemcpyHostToDevice);

	if(hipArray == 0){
		cout << "couldn't allocate memory";
	}

	if(error != hipSuccess){
		cout << "Cuda Error: " << hipGetErrorString(error);
	}
}

//to copy memory back and free memory on device
//will tell error
template <class T>
void destroyCUDAMem(T*& hipArray, T*& arr, int size){
	hipError_t error;
	error = hipMemcpy(arr, hipArray, sizeof(T) * size, hipMemcpyDeviceToHost);

	if(error != hipSuccess){
		cout << "Cuda Memory Device to Host Error: " << hipGetErrorString(error) << endl;
	}

	error = hipFree(hipArray);

	if(error != hipSuccess){
		cout << "Cuda Destroy error: " << hipGetErrorString(error) << endl;
	}
}

//templated class that calls the correct kernel. also sets the grid and block based off the size which is a power of 2
template <class T>
void call(const char* name, T* arr, int size){

	T* hipArray;

	dim3 grid(1, 1);
	dim3 block(1, 1);

	if(size > 2048){
		int n = (size / 2) / 1024;
		grid = dim3(n, 1);
		block = dim3(1024, 1);
	}
	else{
		block = dim3(size/2,1);
	}

	createCUDAMem(arr, size, hipArray);

	if(strcmp(name, "bubble") == 0){
		BubbleKernel<T><<<grid, block>>>(size, hipArray);
		hipDeviceSynchronize();
	}
	else if(strcmp(name, "merge") == 0){
		IterMergeKernel<T><<<grid, block>>>(size, hipArray);
		hipDeviceSynchronize();
	}

	hipError_t wrong = hipGetLastError();

	if(wrong != hipSuccess){
		cout << hipGetErrorString(wrong) << endl;
	}

	destroyCUDAMem(hipArray, arr, size);
}

//the types of values that are accepted for the kernel calling function

template void
call<int>(const char* name, int *arr, int size);

template void
	call<double>(const char* name, double *arr, int size);

template void
	call<char>(const char* name, char *arr, int size);

#endif