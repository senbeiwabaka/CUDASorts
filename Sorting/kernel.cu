#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

#include <iostream>

using namespace std;

template <class T>
__global__ void BubbleKernel(int size, T* arr){
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	//run the sorting algorithm  for ARRAY_SIZE^2
	for(int i = 0; i < size; i++)
	{
	  //check if the thread id number is even
	  if((id % 2) == 0 && id + 1 < size )
	    {
	      if(arr[id] > arr[id + 1])
	        {
	          T temp = arr[id + 1];
	          arr[id + 1] = arr[id];
	          arr[id] = temp;
	        }
	    }
	
	 __syncthreads();
	
	  //check if the thread id number is odd
	  if((id % 2) != 0 && id + 1 < size )
	    {
	    if(arr[id] > arr[id + 1])
	      {
	         T temp = arr[id + 1];
	         arr[id + 1] =arr[id];
	         arr[id] = temp;
	      }
	   }
	__syncthreads();
	
	}
}

template <class T>
__device__ void Merge(T* arr, T* results, int l, int r, int u){
	int i,j,k;
	i=l; j=r; k=l;
	while (i<r && j<u) { 
	  if (arr[i]<=arr[j]) {
		  results[k]=arr[i]; 
		  i++;} 
	  else {
		  results[k]=arr[j]; 
		  j++;}
	  k++;
	}
	
	while (i<r) { 
	  results[k]=arr[i]; i++; k++;
	}
	
	while (j<u) { 
	  results[k]=arr[j]; j++; k++;
	}
	for (k=l; k<u; k++) { 
	  arr[k]=results[k]; 
	}
}

//come back to this one
template <class T>
__global__ void IterMergeKernel(int size, T* arr){
    int tid = threadIdx.x;
    int k,u,i;

	T* results = new T[size];
    
    k = 1;
    while(k < size)
    {
        i = 1;
        while(i+k < size)
        {
            u = i+k*2;
            if(u > size)
            {
                u = size+1;
            }
            Merge(arr, results, i, i+k, u);
            i = i+k*2;
        }
        k = k*2;
        __syncthreads();
    }
}

template <class T>
void createCUDAMem(T*& arr, int size, T*& hipArray){
	hipError_t error;

	error = hipMalloc((void**)&hipArray, sizeof(T) * size);

	if(error != hipSuccess)
		cout << "Cuda Error: " << hipGetErrorString(error);

	error = hipMemcpy(hipArray, arr, sizeof(T) * size, hipMemcpyHostToDevice);

	if(hipArray == 0)
		cout << "couldn't allocate memory";

	if(error != hipSuccess)
		cout << "Cuda Error: " << hipGetErrorString(error);
}

template <class T>
void destroyCUDAMem(T*& hipArray, T*& arr, int size){
	hipError_t error;
	error = hipMemcpy(arr, hipArray, sizeof(T) * size, hipMemcpyDeviceToHost);

	if(error != hipSuccess)
		cout << "Cuda Error: " << hipGetErrorString(error);
	error = hipFree(hipArray);
}

template <class T>
void call(const char* name, T* arr, int size){

	T* hipArray;

	dim3 grid(1, 1);
	dim3 block(size / 2, 1);

	createCUDAMem(arr, size, hipArray);

	if(strcmp(name, "bubble") == 0){
		BubbleKernel<T><<<grid, block>>>(size, hipArray);
	}
	else if(strcmp(name, "merge") == 0){
		IterMergeKernel<T><<<grid, block>>>(size, hipArray);
	}

	hipDeviceSynchronize();

	hipError_t wrong = hipGetLastError();

	if(wrong != hipSuccess)
	{
		cout << hipGetErrorString(wrong) << endl;
	}

	destroyCUDAMem(hipArray, arr, size);
}

template void
call<int>(const char* name, int *arr, int size);

template void
	call<double>(const char* name, double *arr, int size);

template void
	call<char>(const char* name, char *arr, int size);

#endif