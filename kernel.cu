
#include <hip/hip_runtime.h>
__device__ void GPUquickSort(int arr[],int l,int r);
__device__ void GPUmerge(int arr[],int tempArr[],int x,int mid,int y);
__device__ void GPUmergeSort(int arr[],int tempArr[],int x,int y);
__device__ void GPUstartMerge(int arr[],int tempArr[],int arrSize);
__device__ void GPUinsertionSort(int arr[],int arrLength);
__device__ void GPUselectionSort(int arr[],int arrSize);

__device__ int Rec(int f);