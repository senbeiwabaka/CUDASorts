#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

#include <iostream>
#include "Test.h"

using namespace std;

template <class T>
__global__ void BubbleKernel(int size, T* arr){
	bool swap = true;
	int x = 0; 
	T temp;
	while(swap){
		swap = false;
		x++;
		for( int i = 0; i < size - x; i++){
			if(arr[i] > arr[i + 1]){
				temp = arr[i];
				arr[i] = arr[i + 1];
				arr[i + 1] = temp;
				swap = true;
			}
		}
	}
}

__global__ void InsertionKernel(){
}

template <class T>
void createCUDAMem(T*& arr, dim3 &block, dim3 &grid, int size, T*& hipArray){
	hipMalloc((void**)&hipArray, sizeof(T) * size);
	hipMemcpy(hipArray, arr, sizeof(T) * size, hipMemcpyHostToDevice);

	if(hipArray == 0)
		cout << "couldn't allocate memory";

	hipError_t error = hipGetLastError();

	if(error != hipSuccess)
		cout << "Cuda Error: " << hipGetErrorString(error);

	cout << sizeof(T) * size << " " << sizeof(arr) << endl;
}

template <class T>
void destroyCUDAMem(T*& hipArray, T* arr, int size){
	hipMemcpy(arr, hipArray, sizeof(T) * size, hipMemcpyDeviceToHost);
	hipFree(&hipArray);
}

template <class T>
void call(const char* name, T* arr, int size){
	for(int i = 0; i < size; ++i){
		cout << arr[i] << endl;
	}

	cout << sizeof(arr) << " " << sizeof(T) << " " << sizeof(name) << endl;

	cout << typeid(arr).name() << " " << typeid(T).name() << endl;

	T* hipArray;

	dim3 block, grid;

	createCUDAMem(arr, block, grid, size, hipArray);

	cout << strcmp(name, "bubble");

	if(strcmp(name, "bubble") == 0){
		BubbleKernel<T><<<1, 1>>>(size, hipArray);
	}

	destroyCUDAMem(hipArray, arr, size);

	cout << "After bubble \n";
	for(int i = 0; i < size; ++i){
		cout << arr[i] << endl;
	}
}

template void
call<int>(const char* name, int *arr, int size);

template void
	call<double>(const char* name, double *arr, int size);

template void
	call<char>(const char* name, char *arr, int size);

#endif