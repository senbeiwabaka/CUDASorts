#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

#include <iostream>
#include "hip/hip_runtime.h"
#include "Test.h"

using namespace std;

template <class T>
__global__ void BubbleKernel(int size, T* arr){
	bool swap = true;
	int x = 0; 
	T temp;
	while(swap){
		swap = false;
		x++;
		for( int i = 0; i < size - x; i++){
			if(arr[i] > arr[i + 1]){
				temp = arr[i];
				arr[i] = arr[i + 1];
				arr[i + 1] = temp;
				swap = true;
			}
		}
	}
}

template <class T>
__global__ void InsertionKernel(int size, T* arr){
	int j;
	T temp;
	for(int i = 1; i < size; ++i){
		j = i;
		while(j > 0 && arr[j - 1] > arr[j]){
			temp = arr[j];
			arr[j] = arr[j - 1];
			arr[j - 1] = temp;
			j--;
		}
	}
}

template <class T>
__global__ void IterMergeKernel(int size, T* arr){
	int inc,left,leftMax,right,rightMax,cur;
	T *temp = new T[size];
	inc = 1;
	while(inc < size){
	    left = 0;
	    right = inc;
	    leftMax = right - 1;
	    rightMax = (leftMax + inc < size) ? leftMax + inc : size - 1;
	    cur = 0;
	    while(cur < size){
	        while(left <= leftMax && right <= rightMax){
	            if(arr[right] < arr[left]){
					temp[cur]=arr[right++];
				}
	            else{
					temp[cur]=arr[left++];
				}
	            cur++;
	        }
	        while(right<=rightMax){
				temp[cur++]=arr[right++];
			}
	        while(left<=leftMax){
				temp[cur++]=arr[left++];
			}
	        left=right;
	        right+=inc;
	        leftMax=right-1;
	        rightMax=(leftMax+inc<size)?leftMax+inc:size-1;
	    }
	    inc*=2;
	    for(int i=0;i<size;i++){
			arr[i]=temp[i];
		}
	}

	delete[] temp;

	printf("hello");
}

template <class T>
__global__ void SelectionKernel(int size, T* arr){
	int index;
	T temp;
	for(int i = 0; i < size; ++i){
		index = i;
		for(int j = i + 1; j < size; ++j){
			if(arr[j] < arr[index]){
				index = j;
			}
		}
		if(index != i){
			temp = arr[i];
			arr[i] = arr[index];
			arr[index] = temp;
		}
	}
}

template <class T>
void createCUDAMem(T*& arr, dim3 &block, dim3 &grid, int size, T*& hipArray){
	hipMalloc((void**)&hipArray, sizeof(T) * size);
	hipMemcpy(hipArray, arr, sizeof(T) * size, hipMemcpyHostToDevice);

	if(hipArray == 0)
		cout << "couldn't allocate memory";

	hipError_t error = hipGetLastError();

	if(error != hipSuccess)
		cout << "Cuda Error: " << hipGetErrorString(error);
}

template <class T>
void destroyCUDAMem(T*& hipArray, T*& arr, int size){
	hipMemcpy(arr, hipArray, sizeof(T) * size, hipMemcpyDeviceToHost);
	hipFree(hipArray);
}

template <class T>
void call(const char* name, T* arr, int size){

	T* hipArray;

	dim3 block, grid;

	createCUDAMem(arr, block, grid, size, hipArray);

	if(strcmp(name, "bubble") == 0){
		BubbleKernel<T><<<1, 1>>>(size, hipArray);
	}
	else if(strcmp(name, "merge") == 0){
		IterMergeKernel<T><<<1, 1>>>(size, hipArray);
	}
	else if(strncmp(name, "selection", 9) == 0){
		SelectionKernel<T><<<1, 1>>>(size, hipArray);
	}
	else if(strncmp(name, "insertion", 9) == 0){
		InsertionKernel<T><<<1, 1>>>(size, hipArray);
	}

	destroyCUDAMem(hipArray, arr, size);
}

template void
call<int>(const char* name, int *arr, int size);

template void
	call<double>(const char* name, double *arr, int size);

template void
	call<char>(const char* name, char *arr, int size);

#endif