#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

#include <iomanip>
#include "Test.h"

template <class T>
__global__ void BubbleKernel(int size, T* arr){
	bool swap = true;
	int x = 0; int temp;
	while(swap){
		swap = false;
		x++;
		for( int i = 0; i < size; i++){
			if(arr[i] > arr[i + 1]){
				temp = arr[i];
				arr[i] = arr[i + 1];
				arr[i + 1] = temp;
				swap = true;
			}
		}
	}
}

__global__ void InsertionKernel(){
}

template <class T>
void call(const char* name, T* arr, int size){
	char *n = new char[strlen(name)+1];
	strcpy(n, name);
	//printf("name %s", n);

	if(name == "bubble"){
		BubbleKernel<T><<<1, 1>>>(size, arr);
	}
}

template void
call<int>(const char* name, int *arr, int size);

template void
	call<float>(const char* name, float *arr, int size);

template void
	call<char>(const char* name, char *arr, int size);

#endif