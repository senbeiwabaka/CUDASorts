#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

#include <iomanip>
#include "Test.h"

template <class T>
__global__ void BubbleKernel(int size, T* arr){
	bool swap = true;
	int x = 0; 
	T temp;
	while(swap){
		swap = false;
		x++;
		for( int i = 0; i < size; i++){
			if(arr[i] > arr[i + 1]){
				temp = arr[i];
				arr[i] = arr[i + 1];
				arr[i + 1] = temp;
				swap = true;
			}
		}
	}
}

__global__ void InsertionKernel(){
}

template <class T>
void createCUDAMem(T* arr, dim3 &block, dim3 &grid, int size, T* hipArray){
	hipMalloc((void**)&hipArray, sizeof(arr) * size);
	hipMemcpy(hipArray, arr, sizeof(arr) * size, hipMemcpyHostToDevice);
}

template <class T>
void destroyCUDAMem(T* hipArray, T* arr, int size){
	hipMemcpy(arr, hipArray, sizeof(arr) * size, hipMemcpyDeviceToHost);
	hipFree(hipArray);
}

template <class T>
void call(const char* name, T* arr, int size){
	char *n = new char[strlen(name)+1];
	strcpy(n, name);
	printf("name %s", n);

	T* hipArray;

	dim3 block, grid;

	createCUDAMem(arr, block, grid, size, hipArray);

	if(name == "bubble"){
		BubbleKernel<T><<<1, 1>>>(size, hipArray);
	}

	destroyCUDAMem(hipArray, arr, size);
}

template void
call<int>(const char* name, int *arr, int size);

template void
	call<double>(const char* name, double *arr, int size);

template void
	call<char>(const char* name, char *arr, int size);

#endif