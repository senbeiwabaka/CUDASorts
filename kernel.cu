#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

#include <iostream>
#include "hip/hip_runtime.h"
#include "Test.h"

using namespace std;

template <class T>
__global__ void BubbleKernel(int size, T* arr){
	bool swap = true;
	int x = 0; 
	T temp;
	while(swap){
		swap = false;
		x++;
		for( int i = 0; i < size - x; i++){
			if(arr[i] > arr[i + 1]){
				temp = arr[i];
				arr[i] = arr[i + 1];
				arr[i + 1] = temp;
				swap = true;
			}
		}
	}
}

__global__ void InsertionKernel(){
}

template <class T>
__global__ void IterMergeKernel(int size, T* arr){
	int inc,left,leftMax,right,rightMax,cur;
	T *temp = new T[size];
	inc = 1;
	while(inc < size){
	    left = 0;
	    right = inc;
	    leftMax = right - 1;
	    rightMax = (leftMax + inc < size) ? leftMax + inc : size - 1;
	    cur = 0;
	    while(cur < size){
	        while(left <= leftMax && right <= rightMax){
	            if(arr[right] < arr[left]){
					temp[cur]=arr[right++];
				}
	            else{
					temp[cur]=arr[left++];
				}
	            cur++;
	        }
	        while(right<=rightMax){
				temp[cur++]=arr[right++];
			}
	        while(left<=leftMax){
				temp[cur++]=arr[left++];
			}
	        left=right;
	        right+=inc;
	        leftMax=right-1;
	        rightMax=(leftMax+inc<size)?leftMax+inc:size-1;
	    }
	    inc*=2;
	    for(int i=0;i<size;i++){
			arr[i]=temp[i];
		}
	}

	delete[] temp;
}

template <class T>
void createCUDAMem(T*& arr, dim3 &block, dim3 &grid, int size, T*& hipArray){
	hipMalloc((void**)&hipArray, sizeof(T) * size);
	hipMemcpy(hipArray, arr, sizeof(T) * size, hipMemcpyHostToDevice);

	if(hipArray == 0)
		cout << "couldn't allocate memory";

	hipError_t error = hipGetLastError();

	if(error != hipSuccess)
		cout << "Cuda Error: " << hipGetErrorString(error);
}

template <class T>
void destroyCUDAMem(T*& hipArray, T* arr, int size){
	hipMemcpy(arr, hipArray, sizeof(T) * size, hipMemcpyDeviceToHost);
	hipFree(&hipArray);
}

template <class T>
void call(const char* name, T* arr, int size){

	T* hipArray;

	dim3 block, grid;

	createCUDAMem(arr, block, grid, size, hipArray);

	if(strcmp(name, "bubble") == 0){
		BubbleKernel<T><<<1, 1>>>(size, hipArray);
	}
	else if(strcmp(name, "merge") == 0){
		IterMergeKernel<T><<<1, 1>>>(size, hipArray);
	}

	destroyCUDAMem(hipArray, arr, size);
}

template void
call<int>(const char* name, int *arr, int size);

template void
	call<double>(const char* name, double *arr, int size);

template void
	call<char>(const char* name, char *arr, int size);

#endif